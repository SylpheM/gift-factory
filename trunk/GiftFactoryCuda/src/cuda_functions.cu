#include "hip/hip_runtime.h"
#include "GL/glew.h"
#include "GL/glu.h"

#ifdef _WIN32
#include "CUDA/hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "cuda_functions.h"
#include "TextureManager.hpp"


// Kernel that executes on the CUDA device
__global__ void cuda_render(Flake * vFlakes, unsigned int iYMax, unsigned int izMax)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(vFlakes[index].y < - (float)iYMax/2.f)
	{
		vFlakes[index].y += iYMax;
	}
	vFlakes[index].y -= (float) izMax/1000.f;
}

SnowManager::SnowManager(const unsigned int & nbFlakes, const int & xMax, const int & yMax, const int & zMax)
: m_uiNbFlakes(nbFlakes)
, m_iXMax(xMax)
, m_iYMax(yMax)
, m_iZMax(zMax)
, m_Size (17.f)
, m_Vent(0)
, cameraLastPos(NULL)
, textureId(0)
{
	if( m_uiNbFlakes > 0)
	{
		m_vFlakes = new Flake[m_uiNbFlakes];
		for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
		{
			Flake f;
			//At the start, random position
			f.x = rand()%m_iXMax - (float)m_iXMax/2.f + (float) (rand()%100/100.f);
			f.y = rand()%m_iYMax - (float)m_iYMax/2.f + (float)m_iYMax/3.f + (float) (rand()%100/100.f);
			f.z = - rand()%m_iZMax + 1.f + (float) (rand()%100/100.f);

			m_vFlakes[i] = f;
		}
	}
}

void SnowManager::init()
{
	glGenTextures(1, &textureId);
	glBindTexture(GL_TEXTURE_2D, textureId);
	
	TextureManager::Texture * tex = NULL;
	tex = TextureManager::addAndLoadTexture("bouboule.png", "");
	
	if (tex->texPicture)
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, tex->texPicture->w, tex->texPicture->h, 0, GL_RGBA, GL_UNSIGNED_BYTE, tex->texPicture->pixels);
	}
	else
		std::cout<<"Problem when loading 'bouboule.png'"<<std::endl;

	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

	float fMaxSize = 40.0f;
	float coeffs[] = { 1.0f, .0f, 0.0001f };
    glPointParameterfv(GL_POINT_DISTANCE_ATTENUATION, coeffs);
    
    //Taille du point
    glPointParameterf (GL_POINT_SIZE_MAX, fMaxSize);
    glPointParameterf (GL_POINT_SIZE_MIN, 1.0f);
}

void SnowManager::moveFlakes()
{

}

void SnowManager::update(const float* posCamera)
{
	if(cameraLastPos == NULL)
	{
		cameraLastPos = new GLfloat[3];
		cameraLastPos[0] = 0.f;
		cameraLastPos[1] = 0.f;
		cameraLastPos[2] = 0.f;
	}

	/*for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		//m_vFlakes[i].x += posCamera[0] - cameraLastPos[0];
		//m_vFlakes[i].y -= posCamera[1] - cameraLastPos[1];
		//m_vFlakes[i].z += posCamera[2] - cameraLastPos[2];

		m_vFlakes[i].x -= posCamera[0] - cameraLastPos[0];
		m_vFlakes[i].y += posCamera[1] - cameraLastPos[1];
		//m_vFlakes[i].z -= posCamera[2] - cameraLastPos[2];
	}*/

	for(unsigned int i = 0; i < 3; ++i)
	{
		cameraLastPos[i] = posCamera[i];
	}

	//Flakes position updating
	int block_size = 4;
	int n_blocks = m_uiNbFlakes/block_size + (m_uiNbFlakes%block_size == 0 ? 0:1);

	Flake *a_d;
	size_t size = m_uiNbFlakes * sizeof(Flake);
	hipMalloc((void **) &a_d, size);
	hipMemcpy(a_d, m_vFlakes, size, hipMemcpyHostToDevice);
	cuda_render <<< n_blocks, block_size >>> ((Flake *)a_d, m_iYMax, m_iZMax);
	hipMemcpy(m_vFlakes, a_d, sizeof(Flake)*m_uiNbFlakes, hipMemcpyDeviceToHost);
	hipFree(a_d);

	glUseProgram(0);
	//glEnable(GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_ONE, GL_ONE);
	glPointSize(m_Size);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureId);
	glTexEnvf (GL_POINT_SPRITE_ARB, GL_COORD_REPLACE_ARB, GL_TRUE);

	// Active le remplacement du point par la texture
	glEnable (GL_POINT_SPRITE_ARB);

	glEnable(GL_ALPHA_TEST);
	glAlphaFunc(GL_GREATER, 0.0f);

	//glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
	glBegin(GL_POINTS);
	for(int i = 0; i < m_uiNbFlakes; ++i)
	{
		glVertex3f(m_vFlakes[i].x, m_vFlakes[i].y, m_vFlakes[i].z);
	}
	glEnd();

	// D�sactive le remplacement du point par la texture
	glDisable (GL_POINT_SPRITE_ARB);
	glDisable(GL_TEXTURE_2D);
	glDisable(GL_ALPHA_TEST);
	//glDisable(GL_POINT_SMOOTH);
}