#include "hip/hip_runtime.h"
#include "../../Dependencies/include/CUDA/hip/hip_runtime.h"
//#include <CUDA/hip/hip_runtime.h>

#include "cuda_functions.h"

// Kernel that executes on the CUDA device
__global__ void cuda_render(int lol)
{}

void launch_bb()
{
	dim3 dimGrid(1000 / 256, 1);
	dim3 dimBlock(256);
	cuda_render <<< dimGrid, dimBlock >>> (5);

	std::cout<<"Coucou, chez moi j'ai CUDA !!!"<<std::endl;
}