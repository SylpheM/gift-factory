#include "hip/hip_runtime.h"
#include "GL/glew.h"
#include "GL/glu.h"

#ifdef _WIN32
#include "CUDA/hip/hip_runtime.h"
#else
#include <hip/hip_runtime.h>
#endif

#include "cuda_functions.h"
#include "TextureManager.hpp"


// Kernel that executes on the CUDA device
__global__ void cuda_render(Flake * vFlakes, GLfloat actualAimX, GLfloat lastAimX, unsigned int iXMax, unsigned int iYMax, unsigned int izMax)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if(vFlakes[index].y < - (float)iYMax/2.f)
	{
		vFlakes[index].y += iYMax;
	}

	if(vFlakes[index].x < - (float)iXMax/2.f)
	{
		vFlakes[index].x += iXMax;
	}
	else if(vFlakes[index].x > (float)iXMax/2.f)
	{
		vFlakes[index].x -= iXMax;
	}

	if(lastAimX != 0.f)
	{
		vFlakes[index].x -= actualAimX - lastAimX;

		//vFlakes[index].y += posCamera[1] - cameraLastPos[1];
		//vFlakes[index].z += posCamera[2] - cameraLastPos[2];

		//vFlakes[index].x -= posCamera[0] - cameraLastPos[0];
		//vFlakes[index].y += posCamera[1] - cameraLastPos[1];
		//vFlakes[index].z -= posCamera[2] - cameraLastPos[2];
	}	

	vFlakes[index].y -= (float) izMax/1000.f;
}

SnowManager::SnowManager(const unsigned int & nbFlakes, const int & xMax, const int & yMax, const int & zMax)
: m_uiNbFlakes(nbFlakes)
, m_iXMax(xMax)
, m_iYMax(yMax)
, m_iZMax(zMax)
, m_Size (12.f)
, m_Vent(0)
, cameraLastPos(NULL)
, textureId(0)
{
	if( m_uiNbFlakes > 0)
	{
		m_vFlakes = new Flake[m_uiNbFlakes];
		for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
		{
			Flake f;
			//At the start, random position
			f.x = rand()%m_iXMax - (float)m_iXMax/2.f + (float) (rand()%100/100.f);
			f.y = rand()%m_iYMax - (float)m_iYMax/2.f + (float)m_iYMax/3.f + (float) (rand()%100/100.f);
			f.z = - rand()%m_iZMax + 0.f + (float) (rand()%100/100.f);

			m_vFlakes[i] = f;
		}
	}
}

void SnowManager::init()
{
	glGenTextures(1, &textureId);
	glBindTexture(GL_TEXTURE_2D, textureId);
	
	TextureManager::Texture * tex = NULL;
	tex = TextureManager::addAndLoadTexture("bouboule.png", "");
	
	if (tex->texPicture)
	{
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, tex->texPicture->w, tex->texPicture->h, 0, GL_RGBA, GL_UNSIGNED_BYTE, tex->texPicture->pixels);
	}
	else
		std::cout<<"Problem when loading 'bouboule.png'"<<std::endl;

	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);

	float fMaxSize = 12.0f;
	float coeffs[] = { 1.0f, .0f, 0.005f };
    glPointParameterfv(GL_POINT_DISTANCE_ATTENUATION, coeffs);
    
    //Taille du point
    glPointParameterf (GL_POINT_SIZE_MAX, fMaxSize);
    glPointParameterf (GL_POINT_SIZE_MIN, 1.0f);
}

void SnowManager::moveFlakes()
{

}

void SnowManager::update(const float* posCamera)
{
	if(cameraLastPos == NULL)
	{
		cameraLastPos = new GLfloat[3];
		cameraLastPos[0] = 0.f;
		cameraLastPos[1] = 0.f;
		cameraLastPos[2] = 0.f;
	}

	//Flakes position updating
	int block_size = 4;
	int n_blocks = m_uiNbFlakes/block_size + (m_uiNbFlakes%block_size == 0 ? 0:1);

	Flake *a_flakes;
	//GLfloat a_cameraActualAim[3];
	//GLfloat a_cameraLastAim[3];

	size_t size = m_uiNbFlakes * sizeof(Flake);
	hipMalloc((void **) &a_flakes, size);
	//hipMalloc((void **) &a_cameraActualAim, 3*sizeof(GLfloat));
	//hipMalloc((void **) &a_cameraLastAim, 3*sizeof(GLfloat));

	hipMemcpy(a_flakes, m_vFlakes, size, hipMemcpyHostToDevice);
	//hipMemcpy(a_cameraActualAim, posCamera, 3*sizeof(GLfloat), hipMemcpyHostToDevice);
	//hipMemcpy(a_cameraLastAim, cameraLastPos, 3*sizeof(GLfloat), hipMemcpyHostToDevice);

	cuda_render <<< n_blocks, block_size >>> ((Flake *)a_flakes, posCamera[0], cameraLastPos[0], m_iXMax, m_iYMax, m_iZMax);

	hipMemcpy(m_vFlakes, a_flakes, sizeof(Flake)*m_uiNbFlakes, hipMemcpyDeviceToHost);

	hipFree(a_flakes);
	//hipFree(a_cameraActualAim);
	//hipFree(a_cameraLastAim);

	for(unsigned int i = 0; i < 3; ++i)
	{
		cameraLastPos[i] = posCamera[i];
	}

	//----- Drawing ----
	glUseProgram(0);
	//glEnable(GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_ONE, GL_ONE);
	glPointSize(m_Size);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureId);
	glTexEnvf (GL_POINT_SPRITE_ARB, GL_COORD_REPLACE_ARB, GL_TRUE);

	// Active le remplacement du point par la texture
	glEnable (GL_POINT_SPRITE_ARB);

	glEnable(GL_ALPHA_TEST);
	glAlphaFunc(GL_GREATER, 0.0f);

	//glColor4f(1.0f, 1.0f, 1.0f, 1.0f);
	glBegin(GL_POINTS);
	for(int i = 0; i < m_uiNbFlakes; ++i)
	{
		glVertex3f(m_vFlakes[i].x, m_vFlakes[i].y, m_vFlakes[i].z);
	}
	glEnd();

	// D�sactive le remplacement du point par la texture
	glDisable (GL_POINT_SPRITE_ARB);
	glDisable(GL_TEXTURE_2D);
	glDisable(GL_ALPHA_TEST);
	//glDisable(GL_POINT_SMOOTH);
}