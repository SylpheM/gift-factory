#include "hip/hip_runtime.h"
#include "GL/glew.h"
#include "CUDA/hip/hip_runtime.h"
#include "GL/glu.h"

#include "cuda_functions.h"

/*#include "GL/glu.h"

#include "TextureManager.hpp"
#include "ShaderManager.hpp"

// Kernel that executes on the CUDA device
__global__ void cuda_render(Flake * lol)
{
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	//lol[0].y -= 0.001;
}

SnowManager::SnowManager(const unsigned int & nbFlakes, const int & xMax, const int & yMax, const int & zMax)
: m_uiNbFlakes(nbFlakes)
, m_iXMax(xMax)
, m_iYMax(yMax)
, m_iZMax(zMax)
, m_uiShaderId(0)
, textureId(0)
, m_oFlakes(NULL)
{
	if( m_uiNbFlakes > 0)
	{
		m_vFlakes = new Flake[m_uiNbFlakes];
		m_flakesvertices = new GLfloat[m_uiNbFlakes* 4];
		m_flakesindices = new GLuint[m_uiNbFlakes];
		for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
		{
			Flake f;
			//At the start, random position
			f.x = 0.f - (float)i/10;
			f.y = 0.f;
			f.z = -1.f;

			m_flakesvertices[i*4 + 0] = f.x;
			m_flakesvertices[i*4 + 1] = f.y;
			m_flakesvertices[i*4 + 2] = f.z;
			m_flakesvertices[i*4 + 3] = 1.f;
			std::cout<<"coord : ("<<m_flakesvertices[i*4 + 0]<<","<<m_flakesvertices[i*4 + 1]<<","<<m_flakesvertices[i*4 + 2]<<")"<<std::endl;
			m_flakesindices[i] = i;

			f.size = 10;
			m_vFlakes[i] = f;
		}
	}
}

void SnowManager::init()
{
	// Creates a VBO id for a VBO to store the vertices
    //glGenBuffers(1, &vboId);
	//glGenBuffers(1, &indiceId);
    //glGenBuffers(1, &(this->colorsVboId));
    
    //glBindBuffer(GL_ARRAY_BUFFER, this->vboId);
	//glBufferData(GL_ARRAY_BUFFER, m_uiNbFlakes*4*sizeof(GLfloat), m_flakesvertices, GL_DYNAMIC_DRAW);
	//glBindBuffer(GL_ARRAY_BUFFER, 0);

	//glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, this->indiceId);
	//glBufferData(GL_ELEMENT_ARRAY_BUFFER, m_uiNbFlakes*sizeof(GLuint), m_flakesindices, GL_STATIC_DRAW);
	//glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

	m_oFlakes = new Object("", true, "star", false);
	
	//-------

	//Shader initialization
	char * m_sShaderName = "star";
	ShaderManager * shaderM = ShaderManager::getInstance();
	m_uiShaderId = shaderM->getShaderProgramId(m_sShaderName);
	if(m_uiShaderId == ERROR_VALUE)
	{
		//false if there are only vertex and fragment shader, or true if there are vertex, fragment and geometry shader
		m_uiShaderId = shaderM->addShaders(m_sShaderName, false);
		std::cout<<"new shader"<<std::endl;
	}

	//-----

	
	// Init the texture data
	TextureManager::Texture *tex = new TextureManager::Texture();
	tex->texFileName = "textures/clouds.ppm";
	tex->shaderUniformName = "";
	// Load the texture image
	SDL_Surface *surf = IMG_Load(tex->texFileName.c_str());
	tex->texPicture = surf;
	
	if(tex->texPicture != NULL)
	{
		std::cout << "loaded : " << tex->texFileName << std::endl;
		glEnable(GL_TEXTURE_2D);
		glGenTextures(1, &textureId);
		glBindTexture(GL_TEXTURE_2D, textureId);

		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
		glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
		//glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, tex->texPicture->w, tex->texPicture->h, 0, GL_RGBA, GL_UNSIGNED_BYTE, tex->texPicture->pixels);
	}
	else
	{
		std::cout << "failed to load : " << tex->texFileName << std::endl;
	}
}

void SnowManager::update(const float* posCamera)
{
	//Antialiasing sur les points
	//glEnable(GL_POINT_SMOOTH);
	glEnable(GL_POINT_SPRITE);
	glPointSize (32.f);
	//Gestion de la transparence
	//glEnable(GL_BLEND);
	//glBlendFunc(GL_SRC_ALPHA, GL_ONE_MINUS_SRC_ALPHA);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureId);
	glTexEnvf (GL_POINT_SPRITE_ARB, GL_COORD_REPLACE_ARB, GL_TRUE);
    

	glPushMatrix();
	for(int i = 0; i < 	m_uiNbFlakes; ++i)
	{
		glColor4f(0.f, 0.f, 0.f, 1.0f);
		glBegin (GL_POINTS);
			glVertex3f(posCamera[0] + m_vFlakes[i].x, posCamera[1] + m_vFlakes[i].y, posCamera[2] + m_vFlakes[i].z);
		glEnd();
	}
	glPopMatrix();

	for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		m_flakesvertices[i*4 + 0] = 5;
		m_flakesvertices[i*4 + 1] = 0;
		m_flakesvertices[i*4 + 2] = 0;
		m_flakesvertices[i*4 + 3] = 1.f;
	}
	m_oFlakes->sendVertices(m_flakesvertices, m_uiNbFlakes);
	m_oFlakes->draw(NULL);

	//glUseProgram(m_uiShaderId);
	//glBindBuffer(GL_ARRAY_BUFFER, vboId);
	//glBufferData(GL_ARRAY_BUFFER, m_uiNbFlakes*4*sizeof(GLfloat), m_flakesvertices, GL_DYNAMIC_DRAW);

	//glBindBuffer(GL_ARRAY_BUFFER, this->vboId);
	//glDrawArrays(GL_POINTS, 0, m_uiNbFlakes);
	//glDrawElements(GL_POINTS, m_uiNbFlakes, GL_UNSIGNED_INT, 0);
	//glBindBuffer(GL_ARRAY_BUFFER, 0);

	//glUseProgram(0);
	// D�sactive le remplacement du point par la texture
    glDisable (GL_POINT_SPRITE);
	//glDisable(GL_TEXTURE_2D);
}

void SnowManager::see()
{
	for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		std::cout << m_vFlakes[i].x<< std::endl;
	}
}*/