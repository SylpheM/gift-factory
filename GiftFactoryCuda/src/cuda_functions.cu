#include "hip/hip_runtime.h"
#include "GL/glew.h"
#include "CUDA/hip/hip_runtime.h"
#include "GL/glu.h"

#include "cuda_functions.h"

#include "GL/glu.h"

#include "cuda_functions.h"
#include "TextureManager.hpp"
#include "ShaderManager.hpp"
#include "TGALoader.h"


// Kernel that executes on the CUDA device
__global__ void cuda_render(Flake * lol)
{
	//int index = blockIdx.x * blockDim.x + threadIdx.x;
	lol[0].y -= 0.001;
}

SnowManager::SnowManager(const unsigned int & nbFlakes, const int & xMax, const int & yMax, const int & zMax)
: m_uiNbFlakes(nbFlakes)
, m_iXMax(xMax)
, m_iYMax(yMax)
, m_iZMax(zMax)
, textureId(0)
{
	if( m_uiNbFlakes > 0)
	{
		m_vFlakes = new Flake[m_uiNbFlakes];
		for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
		{
			Flake f;
			//At the start, random position
			f.x = 0.f - (float)i/10;
			f.y = 0.f;
			f.z = -1.f;
			f.size = 12.f;

			m_vFlakes[i] = f;
		}
	}
}

void SnowManager::init()
{
	TGALoader oglt;
	oglt.LoadOpenGLTexture("textures/particle.tga", &textureId, TGA_LINEAR);

	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_REPEAT);
	glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_REPEAT);
	glTexEnvf(GL_TEXTURE_ENV, GL_TEXTURE_ENV_MODE, GL_REPLACE);
}

void SnowManager::update(const float* posCamera)
{
	/*for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		m_vFlakes[i].x = posCamera[0] + (float) i/10;
		m_vFlakes[i].y = posCamera[1] + 0;
		m_vFlakes[i].z = posCamera[2] + -1.f;
	}*/

	//Flakes position updating
	Flake *a_d;
	size_t size = m_uiNbFlakes * sizeof(Flake);
	hipMalloc((void **) &a_d, size);
	hipMemcpy(a_d, m_vFlakes, size, hipMemcpyHostToDevice);
	cuda_render <<< m_uiNbFlakes, 1 >>> ((Flake *)a_d);
	hipMemcpy(m_vFlakes, a_d, sizeof(Flake)*m_uiNbFlakes, hipMemcpyDeviceToHost);
	hipFree(a_d);

	glUseProgram(0);
	//glEnable(GL_POINT_SMOOTH);
	glEnable (GL_BLEND);
	glBlendFunc (GL_ONE, GL_ONE);
	glPointSize(12.f);

	glEnable(GL_TEXTURE_2D);
	glBindTexture(GL_TEXTURE_2D, textureId);
	glTexEnvf (GL_POINT_SPRITE_ARB, GL_COORD_REPLACE_ARB, GL_TRUE);

	// Active le remplacement du point par la texture
	glEnable (GL_POINT_SPRITE_ARB);		

	glBegin(GL_POINTS);
	for(int i = 0; i < m_uiNbFlakes; ++i)
	{
		glVertex3f(m_vFlakes[i].x, m_vFlakes[i].y, m_vFlakes[i].z);
	}
	glEnd();

	// D�sactive le remplacement du point par la texture
	glDisable (GL_POINT_SPRITE_ARB);
	glDisable(GL_TEXTURE_2D);
	//glDisable(GL_POINT_SMOOTH);
}

void SnowManager::see()
{
	for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		std::cout << m_vFlakes[i].x<< std::endl;
	}
}