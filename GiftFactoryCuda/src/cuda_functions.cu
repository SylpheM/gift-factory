#include "hip/hip_runtime.h"
#include "../../Dependencies/include/CUDA/hip/hip_runtime.h"
//#include <CUDA/hip/hip_runtime.h>

#include "cuda_functions.h"

// Kernel that executes on the CUDA device
__global__ void cuda_render(Flake * lol)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	lol[0].x = 5;
}

SnowManager::SnowManager(const unsigned int & nbFlakes, const int & xMax, const int & yMax, const int & zMax)
: m_uiNbFlakes(nbFlakes)
, m_iXMax(xMax)
, m_iYMax(yMax)
, m_iZMax(zMax)
{
	if( m_uiNbFlakes > 0)
	{
		m_vFlakes = new Flake[m_uiNbFlakes];
		for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
		{
			Flake f;
			//At the start, random position
			f.x = 0;
			f.y = 0;
			f.size = 0;
			m_vFlakes[i] = f;
		}
	}
}

void SnowManager::update()
{
	Flake *a_d;  // Pointer to host & device arrays
	size_t size = m_uiNbFlakes * sizeof(Flake);
	hipMalloc((void **) &a_d, size);
	hipMemcpy(a_d, m_vFlakes, size, hipMemcpyHostToDevice);
	cuda_render <<< m_uiNbFlakes, 1 >>> ((Flake *)a_d);
	hipMemcpy(m_vFlakes, a_d, sizeof(Flake)*m_uiNbFlakes, hipMemcpyDeviceToHost);
}

void SnowManager::see()
{
	for(unsigned int i = 0; i < m_uiNbFlakes; ++i)
	{
		std::cout << m_vFlakes[i].x<< std::endl;
	}
}

void launch_bb()
{
	std::cout<<"Coucou, chez moi j'ai CUDA !!!"<<std::endl;
}